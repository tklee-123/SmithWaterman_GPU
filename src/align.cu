#include "hip/hip_runtime.h"
#include "mat.h"
#include <chrono>
#include <fstream>
#include <ios>
#include <iostream>
#include <stdio.h>
#include <string>
#include <vector>
#include <sstream>
#include <thread>
#include <mutex>
#include <algorithm>


#define M 3      // match
#define MM -3    // mismatch
#define W -2     // gap score
#define A_LEN 16 // 16, 32, 64, 256, 1024, 2048, 8192  len of sequence A
#define B_LEN 16 // 16, 32, 64, 256, 1024, 2048, 8192 len of sequence B
#define max(a, b) (((a) > (b)) ? (a) : (b)) // return maximum of two values
#define min(a, b) (((a) < (b)) ? (a) : (b)) // return minimum of two values

// Forward declarations of scoring kernel
__global__ void fill_gpu(Matrix h, Matrix d, char seqA[], char seqB[],
                         const int *k);

// generate random sequence of length n
// Thay thế hàm read_sequence_from_file
void read_sequence_from_file(const std::string& filename, std::vector<char>& seq, int line_number) {
    std::ifstream file(filename);
    if (file.is_open()) {
        std::string line;
        for (int i = 0; i < line_number; ++i) {
            if (!std::getline(file, line)) {
                std::cerr << "Error reading line " << line_number << " from file: " << filename << std::endl;
                file.close();
                return;
            }
        }
        seq.assign(line.begin(), line.end());
        file.close();
    } else {
        std::cerr << "Unable to open file: " << filename << std::endl;
    }
}

std::pair<int, int> fill_cpu(Matrix h, Matrix d, char seqA[], char seqB[]) {

  int full_max_id = 0;
  int full_max_val = 0;

  for (int i = 1; i < h.height; i++) {
    for (int j = 1; j < h.width; j++) {

      // scores
      int max_score = 0;
      int direction = 0;
      int tmp_score;
      int sim_score;

      // comparison positions
      int id = i * h.width + j;                  // current cell
      int abov_id = (i - 1) * h.width + j;       // above cell, 1
      int left_id = i * h.width + (j - 1);       // left cell, 2
      int diag_id = (i - 1) * h.width + (j - 1); // upper-left diagonal cell, 3

      // above cell
      tmp_score = h.elements[abov_id] + W;
      if (tmp_score > max_score) {
        max_score = tmp_score;
        direction = 1;
      }

      // left cell
      tmp_score = h.elements[left_id] + W;
      if (tmp_score > max_score) {
        max_score = tmp_score;
        direction = 2;
      }

      // diagonal cell (preferred)
      char baseA = seqA[j - 1];
      char baseB = seqB[i - 1];
      if (baseA == baseB) {
        sim_score = M;
      } else {
        sim_score = MM;
      }

      tmp_score = h.elements[diag_id] + sim_score;
      if (tmp_score >= max_score) {
        max_score = tmp_score;
        direction = 3;
      }

      // assign scores and direction
      h.elements[id] = max_score;
      d.elements[id] = direction;

      if (max_score > full_max_val) {
        full_max_id = id;
        full_max_val = max_score;
      }
    }
  }
  return std::make_pair(full_max_id, full_max_val);
}

__global__ void fill_gpu(Matrix h, Matrix d, char seqA[], char seqB[],const int k, int max_id_val[]) {

  // scores
  int max_score = 0;
  int direction = 0;
  int tmp_score;
  int sim_score;

  // row and column index depending on anti-diagonal
  int i = threadIdx.x + 1 + blockDim.x * blockIdx.x;
  if (k > A_LEN + 1) {
    i += (k - A_LEN);
  }
  int j = ((k) - i) + 1;

  // comparison positions
  int id = i * h.width + j;
  int abov_id = (i - 1) * h.width + j;       // above cell, 1
  int left_id = i * h.width + (j - 1);       // left cell, 2
  int diag_id = (i - 1) * h.width + (j - 1); // upper-left diagonal cell, 3

  // above cell
  tmp_score = h.elements[abov_id] + W;
  if (tmp_score > max_score) {
    max_score = tmp_score;
    direction = 1;
  }

  // left cell
  tmp_score = h.elements[left_id] + W;
  if (tmp_score > max_score) {
    max_score = tmp_score;
    direction = 2;
  }

  // similarity score for diagonal cell
  char baseA = seqA[j - 1];
  char baseB = seqB[i - 1];
  if (baseA == baseB) {
    sim_score = M;
  } else {
    sim_score = MM;
  }

  // diagonal cell (preferred)
  tmp_score = h.elements[diag_id] + sim_score;
  if (tmp_score >= max_score) {
    max_score = tmp_score;
    direction = 3;
  }

  // assign scores and direction
  h.elements[id] = max_score;
  d.elements[id] = direction;

  // save max score and position
  if (max_score > max_id_val[1]) {
    max_id_val[0] = id;
    max_id_val[1] = max_score;
  }
}

// traceback: starting at the highest score and ending at a 0 score
void traceback(Matrix d, int max_id, char seqA[], char seqB[],
               std::vector<char> &seqA_aligned,
               std::vector<char> &seqB_aligned) {

  int max_i = max_id / d.width;
  int max_j = max_id % d.width;

  // traceback algorithm from maximum score to 0
  while (max_i > 0 && max_j > 0) {

    int id = max_i * d.width + max_j;
    int dir = d.elements[id];

    switch (dir) {
    case 1:
      --max_i;
      seqA_aligned.push_back('-');
      seqB_aligned.push_back(seqB[max_i]);
      break;
    case 2:
      --max_j;
      seqA_aligned.push_back(seqA[max_j]);
      seqB_aligned.push_back('-');
      break;
    case 3:
      --max_i;
      --max_j;
      seqA_aligned.push_back(seqA[max_j]);
      seqB_aligned.push_back(seqB[max_i]);
      break;
    case 0:
      max_i = -1;
      max_j = -1;
      break;
    }
  }
}

// print aligned sequnces
void io_seq(std::vector<char> &seqA_aligned, std::vector<char> &seqB_aligned) {

  std::cout << "Aligned sub-sequences of A and B: " << std::endl;
  int align_len = seqA_aligned.size();
  std::cout << "   ";
  for (int i = 0; i < align_len + 1; ++i) {
    std::cout << seqA_aligned[align_len - i];
  }
  std::cout << std::endl;

  std::cout << "   ";
  for (int i = 0; i < align_len + 1; ++i) {
    std::cout << seqB_aligned[align_len - i];
  }
  std::cout << std::endl << std::endl;
}

// input output function to visualize matrix
void io_score(std::string file, Matrix h, char seqA[], char seqB[]) {
  std::ofstream myfile_tsN;
  myfile_tsN.open(file);

  // print seqA
  myfile_tsN << '\t' << '\t';
  for (int i = 0; i < A_LEN; i++)
    myfile_tsN << seqA[i] << '\t';
  myfile_tsN << std::endl;

  // print vertical seqB on left of matrix
  for (int i = 0; i < h.height; i++) {
    if (i == 0) {
      myfile_tsN << '\t';
    } else {
      myfile_tsN << seqB[i - 1] << '\t';
    }
    for (int j = 0; j < h.width; j++) {
      myfile_tsN << h.elements[i * h.width + j] << '\t';
    }
    myfile_tsN << std::endl;
  }
  myfile_tsN.close();
}

void smith_water_cpu(Matrix h, Matrix d, char seqA[], char seqB[]) {

  // populate scoring and direction matrix and find id of max score
  std::pair<int, int> result = fill_cpu(h, d, seqA, seqB);
  int max_id = result.first;
  // traceback
  std::vector<char> seqA_aligned;
  std::vector<char> seqB_aligned;

  // print aligned sequences
  io_seq(seqA_aligned, seqB_aligned);

  std::cout << std::endl;
  std::cout << "CPU result: " << std::endl;

  // print cpu populated direction and scoring matrix
  io_score(std::string("score.dat"), h, seqA, seqB);
  io_score(std::string("direction.dat"), d, seqA, seqB);
}

void smith_water_gpu(Matrix h, Matrix d, char seqA[], char seqB[]) {
    // allocate and transfer sequence data to device
    char *d_seqA, *d_seqB;
    hipMalloc(&d_seqA, strlen(seqA) * sizeof(char));
    hipMalloc(&d_seqB, strlen(seqB) * sizeof(char));
    hipMemcpy(d_seqA, seqA, strlen(seqA) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_seqB, seqB, strlen(seqB) * sizeof(char), hipMemcpyHostToDevice);

    // initialize matrices for gpu
    int Gpu = 1;
    Matrix d_h(strlen(seqA) + 1, strlen(seqB) + 1, Gpu);
    Matrix d_d(strlen(seqA) + 1, strlen(seqB) + 1, Gpu);
    d_h.load(h, Gpu);
    d_d.load(d, Gpu);

    // max id and value
    int *d_max_id_val;
    std::vector<int> h_max_id_val(2, 0);
    hipMalloc(&d_max_id_val, 2 * sizeof(int));
    hipMemcpy(d_max_id_val, h_max_id_val.data(), 2 * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // loop over diagonals of the matrix
    for (int i = 1; i <= (strlen(seqA) + strlen(seqB) - 1); i++) {
      // i là chỉ số đường chéo, ma trận đang có max là (strlen(seqA) + strlen(seqB) - 1) đường chéo
        int col_idx = max(0, (i - strlen(seqB)));
        // biến này là chỉ số cột đầu tiên của một đường chéo
        int diag_len = min(i, (strlen(seqA) - col_idx));
        // biến này là chiều dài của đường chéo
        // launch the kernel: one block by length of diagonal
        int blks = 1; //1 block
        dim3 dimBlock(diag_len / blks); //kích thước của mỗi block, ở đây mỗi block có số thread bằng số phần tử có trong đường chéo ngược
        dim3 dimGrid(blks);
        fill_gpu<<<dimGrid, dimBlock>>>(d_h, d_d, d_seqA, d_seqB, i, d_max_id_val);
        hipDeviceSynchronize();
    }

    // copy data back
    size_t size = (strlen(seqA) + 1) * (strlen(seqB) + 1) * sizeof(float);
    hipMemcpy(h.elements, d_h.elements, size, hipMemcpyDeviceToHost);
    hipMemcpy(d.elements, d_d.elements, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_max_id_val.data(), d_max_id_val, 2 * sizeof(int), hipMemcpyDeviceToHost);

    // std::cout << "   Max score of " << h_max_id_val[1] << " at " << h_max_id_val[0] << std::endl;
    int max_id = h_max_id_val[0];
    std::vector<char> seqA_aligned;
    std::vector<char> seqB_aligned;
    traceback(d, max_id, seqA, seqB, seqA_aligned, seqB_aligned);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    // visualize output
    // io_seq(seqA_aligned, seqB_aligned);
    // io_score(std::string("score_gpu.dat"), h, seqA, seqB);
    // io_score(std::string("direction_gpu.dat"), d, seqA, seqB);

    // deallocate memory
    d_h.gpu_deallocate();
    d_d.gpu_deallocate();
    hipFree(d_seqA);
    hipFree(d_seqB);
    hipFree(d_max_id_val);
}


char* vectorToCharArray(const std::vector<char>& vec) {
    char* arr = new char[vec.size() + 1]; // +1 để thêm ký tự null
    std::copy(vec.begin(), vec.end(), arr);
    arr[vec.size()] = '\0'; // thêm ký tự null vào cuối mảng
    return arr;
}

int countLines(const std::string& filename) {
  std::ifstream file(filename);
  if (!file.is_open()) {
      std::cerr << "Unable to open file: " << filename << std::endl;
      return -1; // Return -1 to indicate an error
  }

  int lineCount = 0;
  std::string line;
  while (std::getline(file, line)) {
      ++lineCount;
  }

  file.close();
  return lineCount;
}
std::vector<std::string> split(const std::string& s, char delimiter) {
    std::vector<std::string> tokens;
    std::istringstream tokenStream(s);
    std::string token;
    while (std::getline(tokenStream, token, delimiter)) {
        tokens.push_back(token);
    }
    return tokens;
}

std::mutex mutex;

// ... (Không thay đổi các hàm khác)

void process_and_compare_line(const std::string& line, std::vector<char>& seqA, std::vector<std::pair<int, std::string>>& scores) {
    // Split the line into gene sequence and number
    std::vector<std::string> parts = split(line, '\t');

    if (parts.size() == 2) {
        std::vector<char> seqB(parts[0].begin(), parts[0].end());

        char* arrA = vectorToCharArray(seqA);
        char* arrB = vectorToCharArray(seqB);

        // initialize scoring and direction matrices
        Matrix scr_cpu(seqA.size() + 1, seqB.size() + 1); // cpu score matrix
        Matrix dir_cpu(seqA.size() + 1, seqB.size() + 1); // cpu direction
        Matrix scr_gpu(seqA.size() + 1, seqB.size() + 1); // gpu score matrix
        Matrix dir_gpu(seqA.size() + 1, seqB.size() + 1); // gpu direction matrix

        // apply initial condition of 0
        for (int i = 0; i < scr_cpu.height; i++) {
            for (int j = 0; j < scr_cpu.width; j++) {
                int id = i * scr_cpu.width + j;
                scr_cpu.elements[id] = 0;
                dir_cpu.elements[id] = 0;
                scr_gpu.elements[id] = 0;
                dir_gpu.elements[id] = 0;
            }
        }

        // CPU
        std::pair<int, int> result = fill_cpu(scr_cpu, dir_cpu, arrA, arrB);
        int score = result.second;

        std::pair<int, std::string> element(score, parts[1]);
        scores.push_back(element);

        // GPU
        smith_water_gpu(scr_gpu, dir_gpu, arrA, arrB);

        // deallocate memory
        scr_cpu.cpu_deallocate();
        dir_cpu.cpu_deallocate();
        scr_gpu.cpu_deallocate();
        dir_gpu.cpu_deallocate();
    } else {
        std::cerr << "Invalid line format: " << line << std::endl;
    }
}
void read_and_compare_sequences_from_file(const std::string& filename, std::vector<char>& seqA, std::vector<std::pair<int, std::string>>& scores, int num_threads) {
  std::ifstream file(filename);
  if (!file.is_open()) {
      std::cerr << "Error opening file: " << filename << std::endl;
      exit(1);
  }

  std::string line;
  std::vector<std::thread> threads;

  // Process each line along with seqA
  while (std::getline(file, line)) {
      threads.emplace_back(process_and_compare_line, line, std::ref(seqA), std::ref(scores));
  }

  // Wait for all threads to complete
  for (std::thread& t : threads) {
      t.join();
  }

  file.close();
}

int main() {
  std::vector<char> seqA;
  int length = countLines("D:\\Gpu-SW\\src\\dog.txt");
  std::cout << "Số chuỗi trong file tham chiếu:" << length << std::endl;
  //Read seqA from line 1
  read_sequence_from_file("D:\\Gpu-SW\\src\\a.txt", seqA, 1);

  std::vector<std::pair<int, std::string>> scores;
  const int num_threads = 2; // Get the number of available threads
  auto start_time = std::chrono::high_resolution_clock::now();

  read_and_compare_sequences_from_file("D:\\Gpu-SW\\src\\dog.txt", seqA, scores, num_threads);
  std::cout << "Số điểm tương đồng được tính:" << scores.size() << std::endl;
  auto end_time = std::chrono::high_resolution_clock::now();

  // Sort the scores list
  std::sort(scores.begin(), scores.end(), [](const auto& a, const auto& b) {
    return a.first < b.first;  // Compare based on the score (first element of the pair)
  });
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
  std::cout << "Thời gian chạy: " << duration.count() << " microseconds" << std::endl;
  // std::cout << "Scores list: ";

  int maxScore = -1; // Initialize maxScore to a value that is guaranteed to be less than any actual score
  std::string maxNumber;
  for (const auto& element : scores) {
    int score = element.first;
    std::string number = element.second;
    // std::cout << "(" << score << "," << number << ")" << std::endl;
    if (score > maxScore){
      maxScore = score;
      maxNumber = number;
    }
  }
  std::cout << maxNumber << std::endl;

}
